/**
 * Tribus Algo for Denarius
 *
 * tpruvot@github 09 2017 - GPLv3
 *
 */
extern "C" {
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_echo.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "x11/cuda_x11.h"

void jh512_setBlock_80(int thr_id, uint32_t *endiandata);
void jh512_cuda_hash_80(const int thr_id, const uint32_t threads, const uint32_t startNounce, uint32_t *d_hash);
void tribus_echo512_final(int thr_id, uint32_t threads, uint32_t *d_hash, uint32_t *d_resNonce, const uint64_t target);

static uint32_t *d_hash[MAX_GPUS];
static uint32_t *d_resNonce[MAX_GPUS];

// cpu hash

extern "C" void tribus_hash(void *state, const void *input)
{
	uint8_t _ALIGN(64) hash[64];

	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_echo512_context ctx_echo;

	sph_jh512_init(&ctx_jh);
	sph_jh512(&ctx_jh, input, 80);
	sph_jh512_close(&ctx_jh, (void*) hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512(&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512(&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(state, hash, 32);
}

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

// ressources cleanup
extern "C" void free_tribus(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipFree(d_hash[thr_id]);
	hipFree(d_resNonce[thr_id]);

	cuda_check_cpu_free(thr_id);
	init[thr_id] = false;

	hipDeviceSynchronize();
}
