extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

#include <stdio.h>
#include <memory.h>

static uint32_t *d_hash[MAX_GPUS];

// X11 CPU Hash
extern "C" void x11hash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[128] = { 0 };

	// blake1-bmw2-grs3-skein4-jh5-keccak6-luffa7-cubehash8-shavite9-simd10-echo11

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_skein512_context ctx_skein;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;

	sph_blake512_init(&ctx_blake);
	sph_blake512 (&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, (void*) hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*) hash);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512 (&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512 (&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, (void*) hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*) hash);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, (const void*) hash, 64);
	sph_luffa512_close (&ctx_luffa, (void*) hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*) hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512 (&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512 (&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(output, hash, 32);
}

//#define _DEBUG
#define _DEBUG_PREFIX "x11"
#include "cuda_debug.cuh"

static bool init[MAX_GPUS] = { 0 };

