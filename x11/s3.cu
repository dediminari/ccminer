/**
 * S3 Hash (Also called Triple S - Used by 1Coin)
 */

extern "C" {
#include "sph/sph_skein.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

extern void x11_shavite512_setBlock_80(void *pdata);
extern void x11_shavite512_cpu_hash_80(int thr_id, uint32_t threads, uint32_t startNounce, uint32_t *d_hash, int order);

#include <stdint.h>

static uint32_t *d_hash[MAX_GPUS];

/* CPU HASH */
extern "C" void s3hash(void *output, const void *input)
{
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_skein512_context ctx_skein;

	unsigned char hash[64];

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512(&ctx_shavite, input, 80);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512(&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512(&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	memcpy(output, hash, 32);
}

#ifdef _DEBUG
#define TRACE(algo) { \
	if (max_nonce == 1 && pdata[19] <= 1) { \
		uint32_t* debugbuf = NULL; \
		hipHostMalloc(&debugbuf, 32); \
		hipMemcpy(debugbuf, d_hash[thr_id], 32, hipMemcpyDeviceToHost); \
		printf("S3 %s %08x %08x %08x %08x...%08x\n", algo, swab32(debugbuf[0]), swab32(debugbuf[1]), \
			swab32(debugbuf[2]), swab32(debugbuf[3]), swab32(debugbuf[7])); \
		hipHostFree(debugbuf); \
	} \
}
#else
#define TRACE(algo) {}
#endif

static bool init[MAX_GPUS] = { 0 };

