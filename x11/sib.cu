extern "C" {
#include "sph/sph_blake.h"
#include "sph/sph_bmw.h"
#include "sph/sph_groestl.h"
#include "sph/sph_skein.h"
#include "sph/sph_jh.h"
#include "sph/sph_keccak.h"
#include "sph/sph_luffa.h"
#include "sph/sph_cubehash.h"
#include "sph/sph_shavite.h"
#include "sph/sph_simd.h"
#include "sph/sph_echo.h"
#include "sph/sph_streebog.h"
}

#include "miner.h"
#include "cuda_helper.h"
#include "cuda_x11.h"

extern void streebog_cpu_hash_64(int thr_id, uint32_t threads, uint32_t *d_hash);
extern void streebog_hash_64_maxwell(int thr_id, uint32_t threads, uint32_t *d_hash);

#include <stdio.h>
#include <memory.h>

static uint32_t *d_hash[MAX_GPUS];

// Sibcoin CPU Hash
extern "C" void sibhash(void *output, const void *input)
{
	unsigned char _ALIGN(128) hash[128] = { 0 };

	sph_blake512_context ctx_blake;
	sph_bmw512_context ctx_bmw;
	sph_groestl512_context ctx_groestl;
	sph_skein512_context ctx_skein;
	sph_jh512_context ctx_jh;
	sph_keccak512_context ctx_keccak;
	sph_gost512_context ctx_gost;
	sph_luffa512_context ctx_luffa;
	sph_cubehash512_context ctx_cubehash;
	sph_shavite512_context ctx_shavite;
	sph_simd512_context ctx_simd;
	sph_echo512_context ctx_echo;

	sph_blake512_init(&ctx_blake);
	sph_blake512 (&ctx_blake, input, 80);
	sph_blake512_close(&ctx_blake, (void*) hash);

	sph_bmw512_init(&ctx_bmw);
	sph_bmw512 (&ctx_bmw, (const void*) hash, 64);
	sph_bmw512_close(&ctx_bmw, (void*) hash);

	sph_groestl512_init(&ctx_groestl);
	sph_groestl512 (&ctx_groestl, (const void*) hash, 64);
	sph_groestl512_close(&ctx_groestl, (void*) hash);

	sph_skein512_init(&ctx_skein);
	sph_skein512 (&ctx_skein, (const void*) hash, 64);
	sph_skein512_close(&ctx_skein, (void*) hash);

	sph_jh512_init(&ctx_jh);
	sph_jh512 (&ctx_jh, (const void*) hash, 64);
	sph_jh512_close(&ctx_jh, (void*) hash);

	sph_keccak512_init(&ctx_keccak);
	sph_keccak512 (&ctx_keccak, (const void*) hash, 64);
	sph_keccak512_close(&ctx_keccak, (void*) hash);

	sph_gost512_init(&ctx_gost);
	sph_gost512(&ctx_gost, (const void*) hash, 64);
	sph_gost512_close(&ctx_gost, (void*) hash);

	sph_luffa512_init(&ctx_luffa);
	sph_luffa512 (&ctx_luffa, (const void*) hash, 64);
	sph_luffa512_close (&ctx_luffa, (void*) hash);

	sph_cubehash512_init(&ctx_cubehash);
	sph_cubehash512 (&ctx_cubehash, (const void*) hash, 64);
	sph_cubehash512_close(&ctx_cubehash, (void*) hash);

	sph_shavite512_init(&ctx_shavite);
	sph_shavite512 (&ctx_shavite, (const void*) hash, 64);
	sph_shavite512_close(&ctx_shavite, (void*) hash);

	sph_simd512_init(&ctx_simd);
	sph_simd512 (&ctx_simd, (const void*) hash, 64);
	sph_simd512_close(&ctx_simd, (void*) hash);

	sph_echo512_init(&ctx_echo);
	sph_echo512 (&ctx_echo, (const void*) hash, 64);
	sph_echo512_close(&ctx_echo, (void*) hash);

	memcpy(output, hash, 32);
}

//#define _DEBUG
#define _DEBUG_PREFIX "sib"
#include "cuda_debug.cuh"

static bool init[MAX_GPUS] = { 0 };
static bool use_compat_kernels[MAX_GPUS] = { 0 };

